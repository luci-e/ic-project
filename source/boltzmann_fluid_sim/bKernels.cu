#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "bKernels.cuh"
#include "utilities.h"
#include "bSimulator.h"

#include "helper_timer.h"
#include "bKernels.h"

StopWatchInterface* timerk = NULL;

__device__ inline bool inside(long long int x, long long int y, unsigned long long int maxX, unsigned long long int maxY){
	return (x >= 0 && x < maxX && y >= 0 && y < maxY);
}

__global__ void
cudaComputeVelocity(bSimulator* sim) {

	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= sim->dimX || y >= sim->dimY)
		return;

	unsigned long long int elementIdx = y * sim->dimX + x;

	bSimulator::node& n = *(sim->nodes + elementIdx);

	if (n.ntype == bSimulator::nodeType::BASE) {
		float macroVel[2];

		float density = sum(n.newDensities, 9);

		matMul(n.newDensities, sim->speeds, macroVel, 1, 9, 2);
		scalarProd(1.f / density, macroVel, macroVel, 2);
		scalarProd((float)sim->c, macroVel, macroVel, 2);
		n.vel = { macroVel[0], macroVel[1] };
	}
}

__global__ void
cudaComputeEquilibrium(bSimulator* sim) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= sim->dimX || y >= sim->dimY)
		return;

	unsigned long long int elementIdx = y * sim->dimX + x;

	bSimulator::node& n = *(sim->nodes + elementIdx);
	if (n.ntype == bSimulator::nodeType::BASE) {

		float density = sum(n.newDensities, 9);
		float macroVel[2] = { n.vel.x, n.vel.y };

		for (auto j = 0; j < 9; j++) {
			float dotProd = dot(&sim->speeds[2 * j], macroVel, 2);
			n.eqDensities[j] = density * sim->weights[j] * (1.f + 3.f * dotProd / sim->c
				+ 9.f * (pow(dotProd, 2) / sim->csqr) / 2.f
				- 3.f * dot(macroVel, macroVel, 2) / (2.f * sim->csqr));
		}
	}

}

__global__ void
cudaComputeNew(bSimulator* sim) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= sim->dimX || y >= sim->dimY)
		return;

	unsigned long long int elementIdx = y * sim->dimX + x;


	bSimulator::node& n = *(sim->nodes + elementIdx);
	if (n.ntype == bSimulator::nodeType::BASE) {
		float newDensities[9];

		vecSub(n.eqDensities, n.newDensities, newDensities, 9);
		scalarProd((float)sim->viscosity, newDensities, newDensities, 9);
		vecSum(newDensities, n.newDensities, newDensities, 9);
		memcpy(n.densities, newDensities, 9 * sizeof(float));
		memset(n.newDensities, 0.f, 9 * sizeof(float));
	}
}

__global__ void
cudaStream(bSimulator* sim) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= sim->dimX || y >= sim->dimY)
		return;

	unsigned long long int elementIdx = y * sim->dimX + x;
	bSimulator::node& n = *(sim->nodes + elementIdx);


	switch (n.ntype) {
	case bSimulator::nodeType::BASE: {
		for (int j = 0; j < 9; j++) {
			int dx = sim->directions[j][0];
			int dy = sim->directions[j][1];

			if (dx == 0 && dy == 0) {
				n.newDensities[j] = n.densities[j];
				continue;
			}

			int opposite = (j < 5) ? ((j - 1) + 2) % 4 + 1 : ((j - 5) + 2) % 4 + 5;

			long long int newX = n.x + dx;
			long long int newY = n.y + dy;

			if (!inside(x, y, sim->dimX, sim->dimY)) {
				switch (sim->doAtEdge) {

				case bSimulator::edgeBehaviour::LOOP: {
					newX = (newX + sim->dimX) % sim->dimX;
					newY = (newY + sim->dimY) % sim->dimY;

					bSimulator::node& nn = *(sim->nodes + newY * sim->dimX + newX);

					n.newDensities[opposite] += nn.densities[opposite];
					n.densities[j] = 0;
					break;
				}

				case bSimulator::edgeBehaviour::EXIT: {
					n.newDensities[j] = 0;
					break;
				}

				}

				continue;
			}

			bSimulator::node& nn = *(sim->nodes + newY * sim->dimX + newX);

			switch (nn.ntype) {
			case bSimulator::nodeType::BASE: {
				n.newDensities[opposite] += nn.densities[opposite];
				n.densities[j] = 0;
				break;
			}

			case bSimulator::nodeType::WALL: {
				n.newDensities[opposite] += n.densities[j];
				n.densities[j] = 0;
				break;
			}
			}

		}

		break;
	}

	case bSimulator::nodeType::WALL: {

		break;
	}
	}


}

__global__ void
cudaUpdateGraphics(bSimulator* sim)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= sim->dimX || y >= sim->dimY)
		return;

	unsigned long long int elementIdx = y * sim->dimX + x;

	bSimulator::node& n = *(sim->nodes + elementIdx);
	bSimulator::displayNode& dn = *(sim->cudaGLNodes + elementIdx);

	switch (n.ntype) {
	case bSimulator::nodeType::BASE: {
		dn.density = mapNumber<float>(sum(&n.densities[0], 9), 0.f, 1.f, 0.f, 1.f);

		float newSpeeds[2] = { n.vel.x, n.vel.y };
		double mag = magnitude(newSpeeds, 2);

		dn.vel.x = mapNumber<float>(newSpeeds[0] / mag, -1.f, 1.f, 0.f, 1.f);
		dn.vel.y = mapNumber<float>(newSpeeds[1] / mag, -1.f, 1.f, 0.f, 1.f);
		break;
	}

	case bSimulator::nodeType::WALL: {
		dn.density = 1.f;
		dn.vel = { 0,0 };
		break;
	}
	}

}


extern "C" {
	void computeVelocity(bSimulator* sim) {

		sdkCreateTimer(&timerk);
		sdkResetTimer(&timerk);

		sdkStartTimer(&timerk);
		cudaComputeVelocity << < sim->gridDim, sim->blockDim >> > (sim);
		hipDeviceSynchronize();
		sdkStopTimer(&timerk);

		printf("Elapsed velocity time: %f\n", sdkGetTimerValue(&timerk));
	}
	
	void computeEquilibrium(bSimulator* sim){
		sdkCreateTimer(&timerk);
		sdkResetTimer(&timerk);

		sdkStartTimer(&timerk);
		cudaComputeEquilibrium << < sim->gridDim, sim->blockDim >> > (sim);
		hipDeviceSynchronize();
		sdkStopTimer(&timerk);

		printf("Elapsed equilibrium time: %f\n", sdkGetTimerValue(&timerk));

	}
	
	void computeNew(bSimulator* sim){
		sdkCreateTimer(&timerk);
		sdkResetTimer(&timerk);

		sdkStartTimer(&timerk);
		cudaComputeNew << < sim->gridDim, sim->blockDim >> > (sim);
		hipDeviceSynchronize();
		sdkStopTimer(&timerk);

		printf("Elapsed new time: %f\n", sdkGetTimerValue(&timerk));

	}
	
	void stream(bSimulator* sim){
		cudaStream <<< sim->gridDim, sim->blockDim >>> (sim);
		hipDeviceSynchronize();
	}
	void updateGraphics(bSimulator* sim)
	{
		hipGraphicsMapResources(1, &sim->cudaVboNodes, 0);

		hipGraphicsResourceGetMappedPointer((void**)&sim->cudaGLNodes,
			&sim->cudaGLNodesSize,
			sim->cudaVboNodes);


		cudaUpdateGraphics << < sim->gridDim, sim->blockDim >> > (sim);
		hipDeviceSynchronize();

		hipGraphicsUnmapResources(1, &sim->cudaVboNodes, 0);

	}
}
